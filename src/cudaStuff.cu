#include "hip/hip_runtime.h"
#include "headers/cudaStuff.cuh"


__global__ void multi(float *matrix, float *vector, float *resultVector, int lines, int coluns) {

    if(threadIdx.x > lines)
        return;
    
    int index = threadIdx.x*coluns;
    float tempSum = 0;

    for(int colunIdx = 0; colunIdx < coluns; colunIdx++){
        tempSum += matrix[index + colunIdx]*vector[colunIdx];
    }

    resultVector[threadIdx.x] = tanh(tempSum);
}

void CUDA::matrixMultiplication(float *d_matrix, float *d_vector, float *d_resultVector, int lines, int coluns){
    multi <<<1, lines>>> (d_matrix, d_vector, d_resultVector, lines, coluns);
}
